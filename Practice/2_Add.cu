//
//  Add.cu
//  
//
//  Created by Sandeep on 10/1/14.
//
//


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512


__global__ void add(int *a ,int *b , int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x] ;
}

int main(void)
{
int *a,*b,*c;
int *d_a,*d_b,*d_c;
int size = N * sizeof( int ) ; 

hipMalloc( (void**)&d_a,size);
hipMalloc( (void**)&d_b,size);
hipMalloc( (void**)&d_c,size);

a = (int*)malloc( size );
b = (int*)malloc( size );
c = (int*)malloc( size );

for(int i = 0 ; i < N ; i++)
	{
		a[i] = rand();
	}



hipMemcpy( d_a , a , size , hipMemcpyHostToDevice);
hipMemcpy( d_b , b , size , hipMemcpyHostToDevice);

add<<<N,1>>>(d_a,d_b,d_c);

hipMemcpy( c, d_c , size , hipMemcpyDeviceToHost);

for(int i = 0 ;i < N ; i++)
{
printf("%d \n",c[i]);
}

free(a);
free(b);
free(c);

hipFree( d_a );
hipFree( d_b );
hipFree( d_c );


return 0 ; 


}










