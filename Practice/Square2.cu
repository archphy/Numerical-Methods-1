

#include <hip/hip_runtime.h>
#include <stdio.h>

//Function prototype

__global__ void square(int *a , int *b);


int askInput();



int main(void)
{
	int threads = askInput();

	//char * lable1[] = "Enter Number of Threads to be executed ";
	//char * lable2[] = "No of threads = ";
	

	//No of threads 
	int N = threads;

	//Size of array 
	int *a , *b ;
	int *Da , *Db ; 
	int size = N * sizeof(int);


	//Allocate memory CPU
	a = (int*)malloc(size);
	b = (int*)malloc(size);

	//Allocate memory GPU
	hipMalloc((void**)&Da ,size);
	hipMalloc((void**)&Db ,size);

	//Write Data in the CPU array 
	for (int i = 0 ; i < N ; i++)
	{
		a[i] = i;
		// printf("%d",a[i]); 													//DEBUG
	}

	//Copy from CPU to GPU 
	hipMemcpy(Da,a,size,hipMemcpyHostToDevice);

	//Work 
	square<<<1,N>>>(Da,Db);

	//Copy from GPU to CPU
	hipMemcpy(b,Db,size,hipMemcpyDeviceToHost);

	for(int i = 0 ; i < N ; i++)
	{
		printf("%d",b[i]);
		printf(" ");
		if (i % 4 == 0)
		{
			printf("\n");
		}
	}
	printf("\n");

	//Free Memory GPU
	hipFree(Da);
	hipFree(Db);

	//Free Memory CPU
	free(a);
	free(b);


	return 0 ; 

}

__global__ void square(int *a , int *b)
{
	int idx = threadIdx.x;
	b[idx] = a[idx] * a[idx];
}



int askInput()
{
	printf("\n");
	int input;
	printf ("Enter Number of Threads to be executed ");
    scanf ("%d", &input);
    printf ("No of threads = %d", input);
    printf("\n");
    return input;
}














